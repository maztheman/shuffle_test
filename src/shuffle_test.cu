#include "hip/hip_runtime.h"
#include "crypto/blake.hpp"
#include "stratum/primitives/block.hpp"
#include "stratum/streams.hpp"
#include "stratum/arith/uint256.hpp"

#include <cstdint>
#include <ctime>
#include <hip/hip_vector_types.h>

using namespace crypto;

#include "sols.h"

#include "param.h"

typedef unsigned char uchar;
typedef uint64_t ulong;
typedef uint32_t uint;

#define checkCudaErrors(call)								\
do {														\
	hipError_t err = call;									\
	if (hipSuccess != err) {								\
		char errorBuff[512];								\
        sprintf(errorBuff, 			\
			"CUDA error '%s' in func '%s' line %d",			\
			hipGetErrorString(err), __FUNCTION__, __LINE__);	\
		fprintf(stderr, "<error> %s\n", errorBuff); \
		}														\
} while (0)


#define ENCODE_INPUTS(row, slot0, slot1) ((row << 20) | ((slot1 & 0x3ff) << 10) | (slot0 & 0x3ff))
#define DECODE_ROW(REF)   (REF >> 20)
#define DECODE_SLOT1(REF) ((REF >> 10) & 0x3ff)
#define DECODE_SLOT0(REF) (REF & 0x3ff)

typedef struct slot32_s
{
	uint4 x;//16 bytes
	uint4 y;//16 bytes
} slot32_t;


typedef struct row32_s
{
	slot32_t slots[NR_SLOTS];
} row32_t;

typedef struct row16_s
{
	uint4 slots[NR_SLOTS];
} row16_t;

typedef struct row8_s
{
	uint2 slots[NR_SLOTS];
} row8_t;

typedef struct table32_s
{
	row32_t	rows[NR_ROWS];
} table32_t;

typedef struct table16_s
{
	row16_t	rows[NR_ROWS];
} table16_t;

typedef struct table8_s
{
	row8_t	rows[NR_ROWS];
} table8_t;

typedef struct data_s
{
	
	candidate_t		candidates;
	table32_t		round0;
	table32_t		round1;
	table32_t		round2;
	table32_t		round3;
	table32_t		round4;
	table16_t		round5;
	table16_t		round6;
	table16_t		round7;
	table8_t		round8;
	uint			rowCounter0[NR_ROWS];
	uint			rowCounter1[NR_ROWS];
	uint			bin_counter[NR_ROWS * 512];
	uint			sols[1024];
	ulong			blake[16];
} data_t;

__device__ __forceinline__ void rotate32(ulong value, ulong* retval)
{
	uint2* ret = (uint2*)&value;
	uint2* ret2 = (uint2*)retval;
	ret2->y = ret->x;
	ret2->x = ret->y;
}

__device__ __forceinline__ void rotate40(ulong value, ulong* retval)
{
	uint2* ret = (uint2*)&value;
	uint2* ret2 = (uint2*)retval;
	ret2->y = __byte_perm(ret->x, ret->y, 0x2107);
	ret2->x = __byte_perm(ret->x, ret->y, 0x6543);
}

__device__ __forceinline__ void rotate48(ulong value, ulong* retval)
{
	uint2* ret = (uint2*)&value;
	uint2* ret2 = (uint2*)retval;
	ret2->y = __byte_perm(ret->x, ret->y, 0x1076);
	ret2->x = __byte_perm(ret->x, ret->y, 0x5432);
}

__device__ __forceinline__ uint get_lane_id()
{
	uint ret;
	asm volatile("mov.u32 %0, %laneid;" : "=r"(ret));
	return ret;
}

#define rotate(a, bits) ((a) << (bits)) | ((a) >> (64 - (bits)))

#define mix(va, vb, vc, vd, x, y) \
va = (va + vb + x); \
rotate32(va ^ vd, &vd); \
vc = (vc + vd); \
rotate40(vb ^ vc, &vb); \
va = (va + vb + y); \
rotate48(vd ^ va, &vd); \
vc = (vc + vd); \
vb = rotate((vb ^ vc), (ulong)64 - 63);

/*

asm volatile ("{\n\t"
".reg .v4 .u32 v1,v2,v3,v4,v5,v6,v7,v8;\n\t"
"ld.global.v4.u32 v1, [%16];\n\t"
"ld.global.v4.u32 v2, [%16+16];\n\t"
"ld.global.v4.u32 v3, [%16+32];\n\t"
"ld.global.v4.u32 v4, [%16+48];\n\t"
"ld.global.v4.u32 v5, [%16+64];\n\t"
"ld.global.v4.u32 v6, [%16+80];\n\t"
"ld.global.v4.u32 v7, [%16+96];\n\t"
"ld.global.v4.u32 v8, [%16+112];\n\t"
"mov.b64 %0, {v1.x, v1.y};\n\t"
"mov.b64 %1, {v1.z, v1.w};\n\t"
"mov.b64 %2, {v2.x, v2.y};\n\t"
"mov.b64 %3, {v2.z, v2.w};\n\t"
"mov.b64 %4, {v3.x, v3.y};\n\t"
"mov.b64 %5, {v3.z, v3.w};\n\t"
"mov.b64 %6, {v4.x, v4.y};\n\t"
"mov.b64 %7, {v4.z, v4.w};\n\t"
"mov.b64 %8, {v5.x, v5.y};\n\t"
"mov.b64 %9, {v5.z, v5.w};\n\t"
"mov.b64 %10, {v6.x, v6.y};\n\t"
"mov.b64 %11, {v6.z, v6.w};\n\t"
"mov.b64 %12, {v7.x, v7.y};\n\t"
"mov.b64 %13, {v7.z, v7.w};\n\t"
"mov.b64 %14, {v8.x, v8.y};\n\t"
"mov.b64 %15, {v8.z, v8.w};\n\t"
"}\n"
:
"=l"(sv[0]), "=l"(sv[1]), "=l"(sv[2]), "=l"(sv[3]),
"=l"(sv[4]), "=l"(sv[5]), "=l"(sv[6]), "=l"(sv[7]),
"=l"(sv[8]), "=l"(sv[9]), "=l"(sv[10]), "=l"(sv[11]),
"=l"(sv[12]), "=l"(sv[13]), "=l"(sv[14]), "=l"(sv[15])
: "l"(blake_data)
);
*/

/*
__device__ uint cnt0[4096];


template<int SIZE>
__global__
void test(const ulong (&blakey)[SIZE])
{
	ulong v[16];
	v[0] = blakey[0];

	uint idx = blockIdx.x;

	asm volatile (
		"ld.param.u64 %0, [%1];\n"
		: "=l"(v[0]) : "l"(blakey)
		);

	uint cnnnt;
	uint* cnt = cnt0 + idx;

	asm volatile (
		"ldu.global.u32 %0, [%1];\n"
		: "=r"(cnnnt) : "l"(cnt)
		);

}*/

__global__
__launch_bounds__(256, 16)
void kernel_round0(data_t* data, const uint4 *  bla)
{
	uint* rowCounter = data->rowCounter0;
	uint tid = blockIdx.x * blockDim.x + threadIdx.x;//1-1m for inputs
	uint laneid = get_lane_id();
	__shared__ uint4 s_sv[8];

	if (tid < 3) {
		data->candidates.sol_nr[tid] = 0;
	}

	ulong* sv = (ulong*)s_sv;
	ulong v[16];

	if (threadIdx.x < 8) {
		s_sv[threadIdx.x] = bla[threadIdx.x];
	}

	__syncthreads();

	uint4* v_ui4 = (uint4*)v;

	v_ui4[0] = s_sv[0];
	v_ui4[1] = s_sv[1];
	v_ui4[2] = s_sv[2];
	v_ui4[3] = s_sv[3];
	v_ui4[4] = s_sv[4];
	v_ui4[5] = s_sv[5];
	v_ui4[6] = s_sv[6];
	v_ui4[7] = s_sv[7];

	ulong word1 = ((ulong)tid) << 32;

	// round 1
	mix(v[0], v[4], v[8], v[12], 0, word1);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 2
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], word1, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 3
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, word1);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 4
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, word1);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 5
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, word1);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 6
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], word1, 0);
	// round 7
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], word1, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 8
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, word1);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 9
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], word1, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 10
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], word1, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 11
	mix(v[0], v[4], v[8], v[12], 0, word1);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], 0, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);
	// round 12
	mix(v[0], v[4], v[8], v[12], 0, 0);
	mix(v[1], v[5], v[9], v[13], 0, 0);
	mix(v[2], v[6], v[10], v[14], 0, 0);
	mix(v[3], v[7], v[11], v[15], 0, 0);
	mix(v[0], v[5], v[10], v[15], word1, 0);
	mix(v[1], v[6], v[11], v[12], 0, 0);
	mix(v[2], v[7], v[8], v[13], 0, 0);
	mix(v[3], v[4], v[9], v[14], 0, 0);

	// compress v into the blake state; this produces the 50-byte hash
	// (two Xi values)
	v[0] = sv[0] ^ v[0] ^ v[8];
	v[1] = sv[1] ^ v[1] ^ v[9];
	v[2] = sv[2] ^ v[2] ^ v[10];
	v[3] = sv[3] ^ v[3] ^ v[11];
	v[4] = sv[4] ^ v[4] ^ v[12];
	v[5] = sv[5] ^ v[5] ^ v[13];
	v[6] = (sv[6] ^ v[6] ^ v[14]) & 0xffff;

	uint2 rowData[6];
	uint row;
	asm volatile(
		"mov.b64 {%0, %1}, %3;\n"
		"prmt.b32 %2, %0, 0, 17409;\n"
		: "=r"(rowData[0].x), "=r"(rowData[0].y), "=r"(row)
		: "l"(v[0]));

	//0  0,1
	//1  2,3
	//2  4,5
	//3  6.7
		
	asm volatile(
		"mov.b64 {%0, %1}, %2;\n"
		: "=r"(rowData[1].x), "=r"(rowData[1].y) //
		: "l"(v[3])
		);

	row >>= 4;

	uint rowCnt = atomicAdd(rowCounter + row, 1);

	if (rowCnt < 608) {
		slot32_t slot;
		slot.y.w = 0;
		uint r1058;
		asm volatile("prmt.b32 %0, %1, %2, 4660;" : "=r"(r1058) : "r"(rowData[0].x), "r"(rowData[0].y));//0,1
		asm volatile(
			"{\n\t"
			".reg .b32 r1,r2,r3,r4;\n\t"
			"mov.b64 {r1, r2}, %6;\n\t" //2,3
			"mov.b64 {r3, r4}, %7;\n\t" //4,5
			"prmt.b32 %0, r2, r3, 4660;\n\t"  //3,4
			"prmt.b32 %1, %8, r1, 4660;\n\t"  //1,2
			"and.b32 %2, %9, 268435455;\n\t"
			"prmt.b32 %3, r1, r2, 4660;\n\t"  //2,3
			"prmt.b32 %4, r4, %10, 4660;\n\t" //5,6
			"prmt.b32 %5, r3, r4, 4660;\n\t"  //4,5
			"}\n" : "=r"(slot.x.w), "=r"(slot.x.y), "=r"(slot.x.x), "=r"(slot.x.z), "=r"(slot.y.y), "=r"(slot.y.x)
			: "l"(v[1]), "l"(v[2]), "r"(rowData[0].y), "r"(r1058), "r"(rowData[1].x)
			);
		data->round0.rows[row].slots[rowCnt].x = slot.x;
		slot.y.z = tid << 1;
		data->round0.rows[row].slots[rowCnt].y = slot.y;
	}
	/*if (rowCnt < 608) {
		slot32_t slot;
		slot.y.w = 0;

		asm volatile (
			"{\n\t"
			".reg .b32 tt;\n\t"
			"mov.b64 {%0, %1}, %8;\n\t"
			"mov.b64 {%2, %3}, %9;\n\t"
			"prmt.b32 %4, %1, %2, 4660;\n\t"//0x1234
			"prmt.b32 %5, %10, %0, 4660;\n\t"
			"prmt.b32 %6, %0, %1, 4660;\n\t"
			"prmt.b32 tt, %10, %11, 4660;\n\t"
			"and.b32 %7, tt, 268435455;\n\t"
			"}\n"
			: "=r"(rowData[2].x), "=r"(rowData[2].y),
			"=r"(rowData[3].x), "=r"(rowData[3].y),
			"=r"(slot.x.w), "=r"(slot.x.y), "=r"(slot.x.z), "=r"(slot.x.x)
			: "l"(v[1]), "l"(v[2]), "r"(rowData[0].y), "r"(rowData[0].x)
			);

		uint4* slot1 = &data->round0.rows[row].slots[rowCnt].x;

		asm volatile("st.global.v4.u32 [%4], {%0, %1, %2, %3};\n"
			: : "r"(slot.x.x), "r"(slot.x.y), "r"(slot.x.z), "r"(slot.x.w), "l"(slot1)
			);

		asm volatile (
			"prmt.b32 %0, %2, %3, 4660;\n"
			"prmt.b32 %1, %4, %2, 4660;\n"
			: "=r"(slot.y.x), "=r"(slot.y.y)
			: "r"(rowData[3].y), "r"(rowData[1].x), "r"(rowData[3].x)
			);

		slot.y.z = tid * 2;

		data->round0.rows[row].slots[rowCnt].y = slot.y;
	}*/

	asm volatile(
		"prmt.b32 %0, %1, 0, 17426;\n"
		: "=r"(row)
		: "r"(rowData[1].x)
		);

	row >>= 4;

	rowCnt = atomicAdd(rowCounter + row, 1);
	if (rowCnt < 608) {
		slot32_t slot;
		slot.y.w = 0;
		asm volatile(
			"{\n\t"
			".reg .b32 r1, r2, r3, r4, r5, r6, r7;\n\t"
			"prmt.b32 r1, %6, %7, 9029;\n\t"
			"mov.b64 {r2, r3}, %8;\n\t"
			"mov.b64 {r4, r5}, %9;\n\t"
			"prmt.b32 %0, r3, r4, 9029;\n\t"
			"prmt.b32 %1, %7, r2, 9029;\n\t"
			"and.b32 %2, r1, 268435455;\n\t"
			"prmt.b32 %3, r2, r3, 9029;\n\t"
			"mov.b64 {r6, r7}, %10;\n\t"
			"prmt.b32 %4, r5, r6, 9029;\n\t"
			"prmt.b32 %5, r4, r5, 9029;\n\t"
			"}\n" : "=r"(slot.x.w), "=r"(slot.x.y), "=r"(slot.x.x), "=r"(slot.x.z), "=r"(slot.y.y), "=r"(slot.y.x)
			: "r"(rowData[1].x), "r"(rowData[1].y), "l"(v[4]), "l"(v[5]), "l"(v[6])
			);

		data->round0.rows[row].slots[rowCnt].x = slot.x;
		slot.y.z = (tid << 1) + 1;
		data->round0.rows[row].slots[rowCnt].y = slot.y;
		/*asm volatile(
			"{\n\t"
			".reg .b32 tt;\n\t"
			"prmt.b32 tt, %8, %9, 9029;\n\t"
			"mov.b64 {%0, %1}, %10;\n\t"
			"mov.b64 {%2, %3}, %11;\n\t"
			"prmt.b32 %4, %1, %2, 9029;\n\t"
			"prmt.b32 %5, %9, %0, 9029;\n\t"
			"prmt.b32 %6, %0, %1, 9029;\n\t"
			"and.b32 %7, tt, 268435455;\n\t"
			"}\n"
			: "=r"(rowData[4].x), "=r"(rowData[4].y),
			"=r"(rowData[5].x), "=r"(rowData[5].y),
			"=r"(slot.x.w), "=r"(slot.x.y), "=r"(slot.x.z), "=r"(slot.x.x) //slot data to be saved
			: "r"(rowData[1].x), "r"(rowData[1].y),
			"l"(v[4]), "l"(v[5])
			);

		data->round0.rows[row].slots[rowCnt].x = slot.x;

		asm volatile(
			"{\n\t"
			".reg .b32 a,b;\n\t"
			"mov.b64 {a, b}, %2;\n\t"
			"prmt.b32 %0, %4, a, 9029;\n\t"
			"prmt.b32 %1, %3, %4, 9029;\n\t"
			"}\n"
			: "=r"(slot.y.y), "=r"(slot.y.x)
			: "l"(v[6]), "r"(rowData[5].x), "r"(rowData[5].y)
			);

		slot.y.z = (tid * 2) + 1;

		data->round0.rows[row].slots[rowCnt].y = slot.y;*/
	}
}

__global__
__launch_bounds__(608, 16)
void kernel_round1(data_t* data)
{
	__shared__ uint16_t s_collisions[3072];
	__shared__ uint4 s_w0[608];
	__shared__ uint2 s_w1[608];
	__shared__ uint s_cnt[256];
	__shared__ uint s_count;

	uint count;
	uint idx = blockIdx.x;
	uint tid = threadIdx.x;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0) {
		s_count = min(608, data->rowCounter0[idx]);
		data->rowCounter0[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0) {
		count = s_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	for (; tid < 608; tid += blockDim.x) {
		uint4 slot_0;
		uint2 slot_1;
		uint bin = 0;
		uint bin_idx = 0;
		if (tid < count) {
			slot_0 = data->round0.rows[idx].slots[tid].x;
			slot_1 = *(uint2*)&data->round0.rows[idx].slots[tid].y;
			s_w0[tid] = slot_0;
			s_w1[tid] = slot_1;
			bin = slot_0.x >> 20;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(11, cnt);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint16_t* col_ptr = &s_collisions[bin * 12];
			for (uint i = 0; i < bin_idx; i++, col_ptr++) {
				uint16_t col = *col_ptr;
				uint2 o_slot_1 = s_w1[col];
				if (slot_1.y != o_slot_1.y) {
					uint4 o_slot_0 = s_w0[col];
					uint r36 = o_slot_0.x ^ slot_0.x;
					uint r66;
					asm volatile("prmt.b32 %0, %1, 0, 17185;" : "=r"(r66) : "r"(r36));
					uint r67 = r66 & 4095;
					uint row_count = atomicAdd(&data->rowCounter1[r67], 1);
					if (row_count < 608) {
						slot32_t to_slot;
						
						to_slot.x.y = o_slot_0.y ^ slot_0.y;
						to_slot.x.z = o_slot_0.z ^ slot_0.z;
						to_slot.x.w = o_slot_0.w ^ slot_0.w;
						to_slot.x.x = r36 & 255;
						data->round1.rows[r67].slots[row_count].x = to_slot.x;
						to_slot.y.y = o_slot_1.y ^ slot_1.y;
						to_slot.y.x = o_slot_1.x ^ slot_1.x;
						uint r76 = idx << 10;
						uint r77 = col | r76;
						uint r78 = r77 << 10;
						to_slot.y.z = r78 | tid;
						to_slot.y.w = 0;
						data->round1.rows[r67].slots[row_count].y = to_slot.y;
					}
				}
			}
		}
	}
}


__global__
__launch_bounds__(608, 16)
void kernel_round2(data_t* data)
{
	__shared__ uint16_t s_collisions[3072];
	__shared__ uint4 s_w0[608];
	__shared__ uint2 s_w1[608];
	__shared__ uint s_cnt[256];
	__shared__ uint s_row_count;

	//uint* s_cnt = &data->bin_counter[blockIdx.x * 256];

	uint idx = blockIdx.x;
	uint count;
	uint tid = threadIdx.x;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0) {
		s_row_count = min(data->rowCounter1[idx], 608);
		data->rowCounter1[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0) {
		count = s_row_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {
		uint bin_idx = 0;
		uint4 slot_0;
		uint2 slot_1;
		uint bin = 0;

		if (tid < count) {
			slot_0 = data->round1.rows[idx].slots[tid].x;
			bin = slot_0.x;
			slot_1 = *(uint2*)&data->round1.rows[idx].slots[tid].y.x;
			s_w0[tid] = slot_0;
			s_w1[tid] = slot_1;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(cnt, 11);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint16_t* col_ptr = &s_collisions[bin * 12];
			for (uint n = 0; n < bin_idx; n++, col_ptr++) {
				uint16_t col = *col_ptr;
				uint2 o_slot_1 = s_w1[col];
				if (slot_1.y != o_slot_1.y) {
					uint4 o_slot_0 = s_w0[col];
					uint r33 = o_slot_0.y ^ slot_0.y;
					uint r61 = r33 >> 20;
					uint row_count = atomicAdd(&data->rowCounter0[r61], 1);
					if (row_count < 608) {
						slot32_t to_slot;
						to_slot.x.w = o_slot_1.x ^ slot_1.x;
						to_slot.x.y = o_slot_0.z ^ slot_0.z;
						to_slot.x.z = o_slot_0.w ^ slot_0.w;
						to_slot.x.x = r33 & 1048575;
						data->round2.rows[r61].slots[row_count].x = to_slot.x;
						to_slot.y.x = o_slot_1.y ^ slot_1.y;
						uint r69 = idx << 10;
						uint r70 = col | r69;
						uint r71 = r70 << 10;
						to_slot.y.y = r71 | tid;
						to_slot.y.w = to_slot.y.z = 0;
						data->round2.rows[r61].slots[row_count].y = to_slot.y;
					}
				}
			}
		}
	}
}


__global__
__launch_bounds__(608, 16)
void kernel_round3(data_t* data)
{
	__shared__ uint16_t s_collisions[256 * 12];
	__shared__ uint4 s_w0[608];
	__shared__ uint s_w1[608];
	__shared__ uint s_count;

	uint* s_cnt = &data->bin_counter[blockIdx.x * 256];


	uint idx = blockIdx.x;
	uint count;
	uint tid = threadIdx.x;
	
	uint laneid = get_lane_id();
	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0) 
	{
		s_count = min(data->rowCounter0[idx], 608);
		data->rowCounter0[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0)
	{
		count = s_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {

		uint4 slot_0;
		uint  slot_1;
		uint bin = 0;
		uint bin_idx = 0;

		if (tid < count) {
			slot_0 = data->round2.rows[idx].slots[tid].x;
			slot_1 = *(uint*)&data->round2.rows[idx].slots[tid].y;
			s_w0[tid] = slot_0;
			s_w1[tid] = slot_1;
			bin = slot_0.x >> 12;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(cnt, 11);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint16_t* col_ptr = &s_collisions[bin * 12];
			for (uint n = 0; n < bin_idx; n++, col_ptr++) {
				uint16_t col = *col_ptr;
				uint o_slot_1 = s_w1[col];
				if (slot_1 != o_slot_1) {
					uint4 o_slot_0 = s_w0[col];
					uint r54 = o_slot_0.x ^ slot_0.x;
					uint r55 = r54 & 4095;
					uint row_count = atomicAdd(&data->rowCounter1[r55], 1);
					if (row_count < 608) {
						slot32_t to_slot;
						to_slot.x.w = o_slot_1 ^ slot_1;
						to_slot.x.x = o_slot_0.y ^ slot_0.y;
						to_slot.x.y = o_slot_0.z ^ slot_0.z;
						to_slot.x.z = o_slot_0.w ^ slot_0.w;
						data->round3.rows[r55].slots[row_count].x = to_slot.x;
						uint r62 = idx << 10;
						uint r63 = col | r62;
						uint r64 = r63 << 10;
						to_slot.y.x = r64 | tid;
						to_slot.y.y = to_slot.y.z = to_slot.y.w = 0;
						data->round3.rows[r55].slots[row_count].y = to_slot.y;
					}
				}
			}
		}
	}
}


__global__
__launch_bounds__(608, 16)
void kernel_round4(data_t* data)
{
	__shared__ uint16_t s_collisions[256 * 12];
	__shared__ uint4 s_w0[608];
	__shared__ uint s_count;

	uint* s_cnt = &data->bin_counter[blockIdx.x * 256];

	uint idx = blockIdx.x;
	uint count;
	uint tid = threadIdx.x;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0)
	{
		s_count = min(data->rowCounter1[idx], 608);
		data->rowCounter1[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0)
	{
		count = s_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {
		uint4 slot_0;
		uint bin = 0;
		uint bin_idx = 0;

		if (tid < count) {
			slot_0 = data->round3.rows[idx].slots[tid].x;
			s_w0[tid] = slot_0;
			bin = slot_0.x >> 24;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(cnt, 11);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint col_idx = bin * 12;
			for (uint n = 0; n < bin_idx; n++, col_idx++) {
				uint16_t col = s_collisions[col_idx];
				uint4 o_slot_0 = s_w0[col];
				if (o_slot_0.w != slot_0.w) {
					uint r28 = o_slot_0.x ^ slot_0.x;
					uint r52;
					asm volatile("bfe.u32 %0, %1, 12, 12;" : "=r"(r52) : "r"(r28));
					uint row_count = atomicAdd(&data->rowCounter0[r52], 1);
					if (row_count < 608) {
						slot32_t to_slot;
						to_slot.x.w = o_slot_0.w ^ slot_0.w;
						to_slot.x.y = o_slot_0.y ^ slot_0.y;
						to_slot.x.z = o_slot_0.z ^ slot_0.z;
						to_slot.x.x = r28 & 4095;
						data->round4.rows[r52].slots[row_count].x = to_slot.x;
						uint r59 = idx << 10;
						uint r60 = col | r59;
						uint r61 = r60 << 10;
						to_slot.y.x = r61 | tid;
						to_slot.y.y = to_slot.y.z = to_slot.y.w = 0;
						data->round4.rows[r52].slots[row_count].y = to_slot.y;
					}
				}
			}
		}
	}
}

__global__
__launch_bounds__(608, 16)
void kernel_round5(data_t* data)
{
	__shared__ uint16_t s_collisions[3072];
	__shared__ uint4 s_w0[608];
	__shared__ uint s_count;
	
	uint* s_cnt = &data->bin_counter[blockIdx.x * 256];

	uint idx = blockIdx.x;
	uint count;
	uint tid = threadIdx.x;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0)
	{
		s_count = min(data->rowCounter0[idx], 608);
		data->rowCounter0[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0)
	{
		count = s_count;
	}

	__syncthreads();

	__shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {
		uint4 slot_0;
		uint bin = 0;
		uint bin_idx = 0;

		if (tid < count) {
			slot_0 = data->round4.rows[idx].slots[tid].x;
			s_w0[tid] = slot_0;
			bin = slot_0.x >> 4;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(cnt, 11);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint col_idx = bin * 12;
			for (uint n = 0; n < bin_idx; n++, col_idx++) {
				uint16_t col = s_collisions[col_idx];
				uint4 o_slot_0 = s_w0[col];
				if (slot_0.w != o_slot_0.w) {
					uint r28 = o_slot_0.y ^ slot_0.y;
					uint r53 = (o_slot_0.x ^ slot_0.x) & 15;
					uint dst_row;
					asm volatile("prmt.b32 %0, %1, %2, 13063;" : "=r"(dst_row) : "r"(r53), "r"(r28));
					uint row_count = atomicAdd(&data->rowCounter1[dst_row], 1);
					if (row_count < 608) {
						uint4 out_slot;
						out_slot.z = o_slot_0.w ^ slot_0.w;
						out_slot.y = o_slot_0.z ^ slot_0.z;
						uint r60 = idx << 10;
						uint r61 = col | r60;
						uint r62 = r61 << 10;
						out_slot.w = r62 | tid;
						out_slot.x = r28 & 16777215;
						data->round5.rows[dst_row].slots[row_count] = out_slot;
					}
				}
			}
		}
	}
}

__global__
__launch_bounds__(608, 16)
void kernel_round6(data_t* data)
{
	__shared__ uint16_t s_collisions[3072];
	__shared__ uint4 s_w0[608];
	__shared__ uint s_row_count;

	uint* s_cnt = &data->bin_counter[blockIdx.x * 256];

	uint idx = blockIdx.x;
	uint tid = threadIdx.x;
	uint count;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0)
	{
		s_row_count = min(data->rowCounter1[idx], 608);
		data->rowCounter1[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0)
	{
		count = s_row_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {
		uint4 slot_0;
		uint bin = 0;
		uint bin_idx = 0;

		if (tid < count) {
			slot_0 = data->round5.rows[idx].slots[tid];
			s_w0[tid] = slot_0;
			bin = slot_0.x >> 16;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(cnt, 11);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint col_idx = bin * 12;
			for (uint n = 0; n < bin_idx; n++, col_idx++) {
				uint16_t col = s_collisions[col_idx];
				uint4 o_slot_0 = s_w0[col];
				if (slot_0.z != o_slot_0.z) {
					uint r25 = o_slot_0.x ^ slot_0.x;
					uint r50;
					asm volatile("bfe.u32 %0, %1, 4, 12;" : "=r"(r50) : "r"(r25));
					uint row_count = atomicAdd(&data->rowCounter0[r50], 1);

					if (row_count < 608) {
						uint4 to_slot;
						uint r52 = r25 & 15;
						uint r53 = r52 << 4;
						uint r54 = o_slot_0.y ^ slot_0.y;
						uint r55 = r54 >> 28;
						to_slot.z = o_slot_0.z  ^ slot_0.z;
						uint r58 = idx << 10;
						uint r59 = col | r58;
						uint r60 = r59 << 10;
						to_slot.w = r60 | tid;
						to_slot.x = r53 | r55;
						to_slot.y = r54 & 268435455;
						data->round6.rows[r50].slots[row_count] = to_slot;
					}
				}
			}
		}
	}
}

__global__
__launch_bounds__(608, 16)
void kernel_round7(data_t* data)
{
	__shared__ uint16_t s_collisions[3072];
	__shared__ uint4 s_w0[608];
	__shared__ uint s_count;

	uint* s_cnt = &data->bin_counter[blockIdx.x * 256];

	uint idx = blockIdx.x;
	uint count;
	uint tid = threadIdx.x;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0) 
	{
		s_count = min(data->rowCounter0[idx], 608);
		data->rowCounter0[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0)
	{
		count = s_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {
		uint4 slot_0;
		uint bin = 0;
		uint bin_idx = 0;

		if (tid < count) {
			slot_0 = data->round6.rows[idx].slots[tid];
			s_w0[tid] = slot_0;
			bin = slot_0.x;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(cnt, 11);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint16_t* col_ptr = &s_collisions[bin * 12];
			for (uint n = 0; n < bin_idx; n++, col_ptr++) {
				uint16_t col = *col_ptr;
				uint4 o_slot_0 = s_w0[col];
				if (slot_0.z != o_slot_0.z) {
					uint r22 = o_slot_0.y ^ slot_0.y;
					uint r47 = r22 >> 16;
					uint row_count = atomicAdd(&data->rowCounter1[r47], 1);
					if (row_count < 608) {
						uint4 to_slot;
						to_slot.y = o_slot_0.z ^ slot_0.z;
						uint r51 = idx << 10;
						uint r52 = col | r51;
						uint r53 = r52 << 10;
						to_slot.z = r53 | tid;
						to_slot.x = r22 & 65535;
						to_slot.w = 0;
						data->round7.rows[r47].slots[row_count] = to_slot;
					}
				}
			}
		}
	}
}



__global__
__launch_bounds__(608, 16)
void kernel_round8(data_t* data)
{

	__shared__ uint16_t s_collisions[3072];
	__shared__ uint2 s_w0[608];
	__shared__ uint s_cnt[256];
	__shared__ uint s_count;

	//uint* s_cnt = &data->bin_counter[blockIdx.x * 256];

	uint idx = blockIdx.x;
	uint count;
	uint tid = threadIdx.x;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_cnt[tid&255] = 0;
	//}

	if (tid == 0)
	{
		s_count = min(data->rowCounter1[idx], 608); 
		data->rowCounter1[idx] = 0;
	}

	__syncthreads();

	if (laneid == 0)
	{
		count = s_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {
		uint2 slot_0;
		uint bin = 0;
		uint bin_idx = 0;

		if (tid < count) {
			slot_0 = *(uint2*)&data->round7.rows[idx].slots[tid];
			s_w0[tid] = slot_0;
			bin = slot_0.x >> 8;
			uint cnt = atomicAdd(&s_cnt[bin], 1);
			bin_idx = min(cnt, 11);
			s_collisions[bin * 12 + bin_idx] = tid;
		}

		__syncthreads();

		if (bin_idx >= 1) {
			uint16_t* col_ptr = &s_collisions[bin * 12];
			for (uint n = 0; n < bin_idx; n++, col_ptr++) {
				uint16_t col = *col_ptr;
				uint2 o_slot_0 = s_w0[col];
				//printf("%08X %08X\n", slot_0.y, o_slot_0.y);
				if (slot_0.y != o_slot_0.y) {
					uint r40 = o_slot_0.x ^ slot_0.x;
					uint r41 = r40 & 255;
					uint r42 = r41 << 4;
					uint r20 = o_slot_0.y ^ slot_0.y;
					uint r43 = r20 >> 28;
					uint r44 = r42 | r43;
					uint row_count = atomicAdd(&data->rowCounter0[r44], 1);
					if (row_count < 608) {
						uint2 to_slot;
						uint r47 = idx << 10;
						uint r48 = col | r47;
						uint r49 = r48 << 10;
						to_slot.y = r49 | tid;
						to_slot.x = r20 & 268435455;
						data->round8.rows[r44].slots[row_count] = to_slot;
					}
				}
			}
		}
	}
}

__global__
__launch_bounds__(608, 16)
void kernel_round9(data_t* data)
{
	__shared__ uint		s_bincount[256];//256 counters, each index is the value, count is the current index
	__shared__ uint2	s_slot1[608];//first 16 bytes
	__shared__ uint16_t	s_collisions[3072];
	__shared__ uint 	s_count;
	
	uint idx = blockIdx.x;
	uint tid = threadIdx.x;
	uint count;
	uint laneid = get_lane_id();

	//if (tid < 256) {
		s_bincount[tid&255] = 0;//reset bin count
	//}

	if (tid == 0)
	{
		s_count = min(data->rowCounter0[idx], 608);
		data->rowCounter0[idx] = 0;//reset counter after we read it
	}

	__syncthreads();

	if (laneid == 0)
	{
		count = s_count;
	}

	__syncthreads();

	count = __shfl_sync(0xFFFFFFFF, count, 0);

	if (tid > 607) { return; }

	for (; tid < 608; tid += blockDim.x) {
		uint bin_idx = 0;
		uint2 slot1;
		uint bin = 0;

		if (tid < count) {
			slot1 = data->round8.rows[idx].slots[tid];
			s_slot1[tid] = slot1;
			bin = slot1.x >> 20;//top 8 bits of 0xFFFFFFF
			uint cnt = atomicAdd(&s_bincount[bin], 1);
			bin_idx = min(cnt, 11);//something like only 12 collisions please, 0-11 index
			s_collisions[bin * 12 + bin_idx] = tid;//nr_collision_slots is 12, so i can know how many slots are same value because they are being stored in this way
		}

		//binning is done for first 512 slots
		__syncthreads();//because the shared memory writes

		if (bin_idx >= 1) {
			uint col_idx = bin * 12;
			for (uint i = 0; i < bin_idx; i++, col_idx++) {
				uint col = s_collisions[col_idx];
				uint2 other_slot1 = s_slot1[col];
				if (other_slot1.x == slot1.x) {
					uint tmp = other_slot1.y ^ slot1.y;
					bool rc = true;
					if (tmp < 1048576) {//0x1000000
						rc = ((tmp & 1047552) != 0) && ((tmp & 1023) != 0);
					}
					bool rc2 = ((other_slot1.y ^ (slot1.y >> 10)) & 0x3FF) != 0;
					bool rc3 = ((slot1.y ^ (other_slot1.y >> 10)) & 0x3FF) != 0;
					bool rc4 = rc2 && rc3 && rc;
					if (rc4) {
						//increase sol match
						uint cnt = atomicAdd(&data->candidates.sol_nr[2], 1);
						if (cnt < 1024) {
							data->sols[cnt] = ((col | (idx << 10)) << 10) | tid;
						}
					}
				}
			}
		}
	}
}


__global__
__launch_bounds__(128, 16)
void kernel_candidates(data_t* data)
{
	__shared__ uint s_candidate[512];
	__shared__ uint16_t s_test[2048];
	__shared__ uint s_cnts[64];
	__shared__ uint s_is_col;
	__shared__ uint s_sol_num;

	uint tid = threadIdx.x;
	if (!tid) {
		s_is_col = 0;
	}

	for (int i = tid; i < 64; i += blockDim.x) {
		s_cnts[i] = 0;
	}

	uint idx = blockIdx.x;
	uint cnt = data->candidates.sol_nr[2];//r83
	if (idx >= cnt) {
		return;
	}

	if (tid < 512) {
		uint encoded_row = data->sols[idx];
		uint sl_row = encoded_row >> 20;
		uint slot_a = encoded_row >> 10;
		//uint block_count = blockDim.x;
		//uint addr = row * 4864;
		//char* addr_p = (char*)data + (row * 4864);
		for(uint n = tid; n < 512; n += blockDim.x) {
			uint sl_slot = (n < 256 ? slot_a : encoded_row) & 0x3FF;
			//round 8 is 8 bytes
			uint r8_enc = data->round8.rows[sl_row].slots[sl_slot].y;//4
			uint r8_row = r8_enc >> 20;
			uint r8_slot = ((n & 128) < 128 ? (r8_enc >> 10) : r8_enc) & 0x3FF;
			//round 7 is 16 bytes
			uint r7_enc = data->round7.rows[r8_row].slots[r8_slot].z;//8
			uint r7_row = r7_enc >> 20;
			uint r7_slot = ((n & 64) < 64 ? (r7_enc >> 10) : r7_enc) & 0x3FF;
			//round 6 is 16 bytes
			uint r6_enc = data->round6.rows[r7_row].slots[r7_slot].w;//12
			uint r6_row = r6_enc >> 20;
			uint r6_slot = ((n & 32) < 32 ? (r6_enc >> 10) : r6_enc) & 0x3FF;
			//round 5 is 16 bytes ?
			uint r5_enc = data->round5.rows[r6_row].slots[r6_slot].w;//12
			uint r5_row = r5_enc >> 20;
			uint r5_slot = ((n & 16) < 16 ? (r5_enc >> 10) : r5_enc) & 0x3FF;
			//round 4 is 32 bytes ?
			uint r4_enc = data->round4.rows[r5_row].slots[r5_slot].y.x;//16
			uint r4_row = r4_enc >> 20;
			uint r4_slot = ((n & 8) < 8 ? (r4_enc >> 10) : r4_enc) & 0x3FF;
			//round 3 is 32 bytes
			uint r3_enc = data->round3.rows[r4_row].slots[r4_slot].y.x;//16
			uint r3_row = r3_enc >> 20;
			uint r3_slot = ((n & 4) < 4 ? (r3_enc >> 10) : r3_enc) & 0x3FF;
			//round 2 is 32 bytes
			uint r2_enc = data->round2.rows[r3_row].slots[r3_slot].y.y;//20
			uint r2_row = r2_enc >> 20;
			uint r2_slot = ((n & 2) < 2 ? (r2_enc >> 10) : r2_enc) & 0x3FF;
			//round 1 is 32 bytes
			uint r1_enc = data->round1.rows[r2_row].slots[r2_slot].y.z;//24
			uint r1_row = r1_enc >> 20;
			uint r1_slot = (((n & 1) != 1) ? (r1_enc >> 10) : r1_enc) & 0x3FF;
			//round 0 is 32 bytes
			uint r0_enc = data->round0.rows[r1_row].slots[r1_slot].y.z;//24
			s_candidate[n] = r0_enc;
		}
	}

	bool rc = tid < 512;

	__syncthreads();

	for (uint n = tid; n < 512; n += blockDim.x) {
		uint cand = s_candidate[n];
		uint cand_xx = cand & 63;
		uint cnt = atomicAdd(&s_cnts[cand_xx], 1);

		if (cnt < 32) {
			s_test[cand_xx * 32 + cnt] = cand >> 6;
		}
	}

	__syncthreads();

	uint t1 = tid >> 31;
	uint t2 = tid + t1;
	uint t3 = t2 >> 1;
	uint l_cnt = s_cnts[t3];
	l_cnt = min(l_cnt, 32);
	uint cnt2 = l_cnt - 1;
	uint cnt3 = cnt2 * l_cnt;
	uint cnt4 = cnt3 >> 31;
	uint cnt5 = cnt3 + cnt4;
	uint cnt_max = cnt5 >> 1;
	uint t4 = t2 & 4294967294U;
	uint t5 = tid - t4;

	if (t5 < cnt_max) {
		for (uint n = t5; n < cnt_max; n += 2) {
			uint d1 = ((n + (n / l_cnt)) + 1) % l_cnt;
			uint d2 = n % l_cnt;
			uint16_t r1 = s_test[t3 * 32 + d1];
			uint16_t r2 = s_test[t3 * 32 + d2];
			if (r1 == r2) {
				atomicAdd(&s_is_col, 1);
			}
		}
	}

	__syncthreads();

	if (s_is_col != 0) { return; }

	const uint tid_idx = tid * 4;
	uint cand1 = s_candidate[tid_idx + 1];
	uint cand2 = s_candidate[tid_idx];

	if (cand2 > cand1) {
		s_candidate[tid_idx] = cand1;
		s_candidate[tid_idx + 1] = cand2;
	}
	cand1 = s_candidate[tid_idx + 3];
	cand2 = s_candidate[tid_idx + 2];

	if (cand2 > cand1) {
		s_candidate[tid_idx + 2] = cand1;
		s_candidate[tid_idx + 3] = cand2;
	}
	__syncthreads();

	cand1 = s_candidate[tid_idx + 2];
	cand2 = s_candidate[tid_idx];

	if (cand2 > cand1) {
		uint tid_idx2 = tid_idx + 2;
		for (uint n = tid * 4; n < tid_idx2; n++) {
			uint u1 = s_candidate[n + 2];
			s_candidate[n] = u1;
			s_candidate[n + 2] = cand2;
			cand2 = s_candidate[n + 1];
		}
	}

	__syncthreads();

	uint tt3 = t3 * 4;
	uint cand2_1 = s_candidate[tt3 + 4];
	uint cand2_2 = s_candidate[tt3];

	if (cand2_2 > cand2_1) {
		uint n_max = (t5 << 1) + tt3 + 1;
		for (uint n = (t3 * 8 + (t5 << 1)) - 1; n < n_max; n++) {
			uint tr1 = s_candidate[n];
			uint tr2 = s_candidate[n + 4];
			s_candidate[n] = tr2;
			s_candidate[n + 4] = tr1;
		}

	}

	__syncthreads();

	uint r165 = tid >> 31;
	uint r166 = r165 >> 30;
	uint r167 = tid + r166;
	uint r39 = r167 >> 2;
	uint r40 = r39 << 4;
	uint r168 = s_candidate[r40 + 8];
	uint r169 = s_candidate[r40];

	if (r169 > r168) {
		uint r173 = r167 & 2147483644;
		uint r174 = tid - r173;
		uint r175 = r174 << 1;
		uint r176 = r175 + r40;
		uint r41 = r176 + 1;
		uint r177 = r39 * 16 + r175;
		for (uint r269 = r177; r269 < r41; r269++) {
			uint r178 = s_candidate[r269];
			uint r179 = s_candidate[r269 + 8];
			s_candidate[r269] = r179;
			s_candidate[r269 + 8] = r178;
		}
	}

	__syncthreads();

	uint r181 = r165 >> 29;
	uint r182 = tid + r181;
	uint r45 = r182 >> 3;
	uint r46 = r45 << 5;
	uint r183 = s_candidate[r46 + 16];
	uint r184 = s_candidate[r46];

	if (r184 > r183) {
		uint r188 = r182 & 2147483640;
		uint r189 = tid - r188;
		uint r190 = r189 << 1;
		uint r191 = r190 + r46;
		uint r47 = r191 + 1;
		uint r192 = r45 * 32 + r190;
		for (uint r270 = r192; r270 < r47; r270++) {
			uint r193 = s_candidate[r270];
			uint r194 = s_candidate[r270 + 16];
			s_candidate[r270] = r194;
			s_candidate[r270 + 16] = r193;
		}
	}

	__syncthreads();

	uint r196 = r165 >> 28;
	uint r197 = tid + r196;
	uint r51 = r197 >> 4;
	uint r52 = r51 << 6;
	uint r198 = s_candidate[r52 + 32];
	uint r199 = s_candidate[r52];

	if (r199 > r198) {
		uint r203 = r197 & 2147483632;
		uint r204 = tid - r203;
		uint r205 = r204 << 1;
		uint r206 = r205 + r52;
		uint r53 = r206 + 1;
		uint r207 = r51 * 64 + r205;
		for (uint r271 = r207; r271 < r53; r271++) {
			uint r208 = s_candidate[r271];
			uint r209 = s_candidate[r271 + 32];
			s_candidate[r271] = r209;
			s_candidate[r271 + 32] = r208;
		}
	}

	__syncthreads();

	uint r211 = r165 >> 27;
	uint r212 = tid + r211;
	uint r57 = r212 >> 5;
	uint r58 = r57 << 7;
	uint r213 = s_candidate[r58 + 64];
	uint r214 = s_candidate[r58];


	if (r214 > r213) {
		uint r218 = r212 & 2147483616;
		uint r219 = tid - r218;
		uint r220 = r219 << 1;
		uint r221 = r220 + r58;
		uint r59 = r221 + 1;
		uint r222 = r57 * 128 + r220;
		for (uint r272 = r222; r272 < r59; r272++) {
			uint r223 = s_candidate[r272];
			uint r224 = s_candidate[r272 + 64];
			s_candidate[r272] = r224;
			s_candidate[r272 + 64] = r223;
		}
	}

	__syncthreads();

	uint r226 = r165 >> 26;
	uint r227 = tid + r226;
	uint r63 = r227 >> 6;
	uint r64 = r63 << 8;
	uint r228 = s_candidate[r64 + 128];
	uint r229 = s_candidate[r64];

	if (r229 > r228) {
		uint r233 = r227 & 2147483584;
		uint r234 = tid - r233;
		uint r235 = r234 << 1;
		uint r236 = r235 + r64;
		uint r65 = r236 + 1;
		uint r237 = r63 * 256 + r235;
		for (uint r273 = r237; r273 < r65; r273++) {
			uint r238 = s_candidate[r273];
			uint r239 = s_candidate[r273 + 128];
			s_candidate[r273] = r239;
			s_candidate[r273 + 128] = r238;
		}
	}

	__syncthreads();

	uint r241 = r165 >> 25;
	uint r242 = tid + r241;
	uint r69 = r242 >> 7;
	uint r70 = r69 << 9;
	uint r243 = s_candidate[r70 + 256];
	uint r244 = s_candidate[r70];

	if (r244 > r243) {
		uint r248 = r242 & 2147483520;
		uint r249 = tid - r248;
		uint r250 = r249 << 1;
		uint r251 = r250 + r70;
		uint r71 = r251 + 1;
		uint r252 = r69 * 512 + r250;

		for (uint r274 = r252; r274 < r71; r274++) {
			uint r253 = s_candidate[r274];
			uint r254 = s_candidate[r274 + 256];
			s_candidate[r274] = r254;
			s_candidate[r274 + 256] = r253;
		}
	}

	rc = tid == 0;

	__syncthreads();

	if (rc) {
		uint solc = atomicAdd(&data->candidates.sol_nr[0], 1);
		s_sol_num = solc;
	}

	__syncthreads();

	uint solc = s_sol_num;
	if (solc < 16) {
		int r76 = tid_idx + 3;
		uint* p_cand = &data->candidates.vals[solc][tid_idx];
		uint* p_s_cand = &s_candidate[tid_idx];
		for (int r275 = (int)tid_idx - 1; r275 < r76; r275++, p_cand++, p_s_cand++) {
			*p_cand = *p_s_cand;
		}
	}

}

struct context
{
	data_t*			d_data;
	uint4*			d_blake_data;
	candidate_t*	h_candidates;

	void init()
	{
		checkCudaErrors(hipSetDevice(0));
		checkCudaErrors(hipDeviceReset());
		checkCudaErrors(hipMalloc((void**)&d_data, sizeof(data_t)));
		checkCudaErrors(hipMalloc((void**)&d_blake_data, 128));
		checkCudaErrors(hipMemset(d_data, 0, sizeof(data_t)));
		checkCudaErrors(hipHostMalloc(&h_candidates, sizeof(candidate_t)));
	}


	void destroy()
	{
		checkCudaErrors(hipSetDevice(0));
		checkCudaErrors(hipDeviceReset());
		//checkCudaErrors(hipHostFree(h_candidates));
	}
};


#define COLLISION_BIT_LENGTH (PARAM_N / (PARAM_K+1))
#define COLLISION_BYTE_LENGTH ((COLLISION_BIT_LENGTH+7)/8)
#define FINAL_FULL_WIDTH (2*COLLISION_BYTE_LENGTH+sizeof(uint32_t)*(1 << (PARAM_K)))

#define NDIGITS   (PARAM_K+1)
#define DIGITBITS (PARAM_N/(NDIGITS))
#define PROOFSIZE (1u<<PARAM_K)
#define COMPRESSED_PROOFSIZE ((COLLISION_BIT_LENGTH+1)*PROOFSIZE*4/(8*sizeof(uint32_t)))


#include <mutex>
struct speed_test
{
	using time_point = std::chrono::high_resolution_clock::time_point;

	time_point m_start;
	int m_interval;

	speed_test(int interval)
		: m_start(std::chrono::high_resolution_clock::now())
		, m_interval(interval)
	{

	}

	std::vector<time_point> solutions;
	std::mutex sol_mutex;

	void AddSolution() {
		std::lock_guard<std::mutex> l(sol_mutex);
		solutions.push_back(std::chrono::high_resolution_clock::now());
	}

	double GetSolutionSpeed()
	{
		return Get(solutions, sol_mutex);
	}

	double Get(std::vector<time_point>& buffer, std::mutex& mutex)
	{
		time_point now = std::chrono::high_resolution_clock::now();
		time_point past = now - std::chrono::seconds(m_interval);
		double interval = (double)m_interval;
		if (past < m_start)
		{
			interval = ((double)std::chrono::duration_cast<std::chrono::milliseconds>(now - m_start).count()) / 1000;
			past = m_start;
		}
		size_t total = 0;

		mutex.lock();
		for (std::vector<time_point>::iterator it = buffer.begin(); it != buffer.end();)
		{
			if ((*it) < past)
			{
				it = buffer.erase(it);
			}
			else
			{
				++total;
				++it;
			}
		}
		mutex.unlock();

		return (double)total / (double)interval;
	}

};

speed_test speed(300);
std::vector<uint> bin_counter(NR_ROWS * 512);
std::vector<uint> row_counter(NR_ROWS);



int bins[512] = { 0 };

template<int END = 256>
void PrintAverageBinCount(int round, context& ctx)
{
	std::fill(&bin_counter[0], &bin_counter[END], 0);
	checkCudaErrors(hipMemcpy(&bin_counter[0], ctx.d_data->bin_counter, NR_ROWS * END * 4, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	std::fill(&bins[0], &bins[END], 0);

	for (int i = 0; i < NR_ROWS; i++) {
		for (int n = 0; n < END; n++) {
			bins[n] += bin_counter[i * END + n];
		}
	}

	int value = 0;

	for (int i = 0; i < END; i++) {
		value += bins[i];
	}

	int avg = value / END;

	//printf("Round %d: Avg Bin Count: %d\n", round, avg);
}

void PrintAverageRowCount(int round, context& ctx)
{
	std::fill(row_counter.begin(), row_counter.end(), 0);

	if (round % 2 == 0) {
		//row counter 1
		checkCudaErrors(hipMemcpy(&row_counter[0], ctx.d_data->rowCounter1, NR_ROWS * 4, hipMemcpyDeviceToHost));
	} else {
		//row couter 0
		checkCudaErrors(hipMemcpy(&row_counter[0], ctx.d_data->rowCounter0, NR_ROWS * 4, hipMemcpyDeviceToHost));
	}

	hipDeviceSynchronize();

	int cnt = 0;
	for (int i = 0; i < NR_ROWS; i++) {
		cnt += row_counter[i];
	}

	int avg = cnt / NR_ROWS;

	printf("Round %d: Avg Row Count: %d\n", round - 1, avg);


}

/*
struct context_v1
{
	char* d_ht0;
	char* d_ht1;
	char* d_rowCounter0;
	char* d_rowCounter1;
	sols_t* d_sols;
	sols_t* h_sols;

	void init()
	{
		checkCudaErrors(hipSetDevice(0));
		checkCudaErrors(hipDeviceReset());

		checkCudaErrors(hipMalloc((void**)&d_ht0, HT_SIZE));
		checkCudaErrors(hipMalloc((void**)&d_ht1, HT_SIZE));

		checkCudaErrors(hipMalloc((void**)&d_rowCounter0, NR_ROWS));
		checkCudaErrors(hipMalloc((void**)&d_rowCounter1, NR_ROWS));

		checkCudaErrors(hipMalloc((void**)&d_sols, sizeof(sols_t)));
		checkCudaErrors(hipHostMalloc((void**)&h_sols, sizeof(sols_t)));
	}

	void destroy()
	{
		checkCudaErrors(hipFree(d_ht0));
		checkCudaErrors(hipFree(d_ht1));
		checkCudaErrors(hipFree(d_rowCounter0));
		checkCudaErrors(hipFree(d_rowCounter1));
		checkCudaErrors(hipFree(d_sols));
		checkCudaErrors(hipHostFree(h_sols));
	}
};

static void solve_v1(context_v1& ctx, const char* header, unsigned int header_len, const char* nonce, unsigned int nonce_len)
{
	unsigned char mcontext[140];
	memset(mcontext, 0, 140);
	memcpy(mcontext, header, header_len);
	memcpy(mcontext + header_len, nonce, nonce_len);

	blake2b_state_t initialCtx;
	zcash_blake2b_init(&initialCtx, ZCASH_HASH_LEN, PARAM_N, PARAM_K);
	zcash_blake2b_update(&initialCtx, (const uint8_t*)mcontext, 128, 0);

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_blake), &initialCtx, sizeof(blake2b_state_s), 0, hipMemcpyHostToDevice));

	char* d_ht0 = ctx.d_ht0;
	char* d_ht1 = ctx.d_ht1;
	char* d_rowCounter0 = ctx.d_rowCounter0;
	char* d_rowCounter1 = ctx.d_rowCounter1;
	sols_t* d_sols = ctx.d_sols;
	sols_t* h_sols = ctx.h_sols;


	kernel_init_v1 << <NR_ROWS / ROWS_PER_UINT / 256, 256 >> > (d_rowCounter0);
	kernel_round0_v1 << < NR_INPUTS / 256, 256 >> > (d_ht0, d_rowCounter0);
}
*/

static void solve(context& ctx, const char* header, unsigned int header_len, const char* nonce, unsigned int nonce_len)
{
	uint64_t blake_data[16];
	unsigned char mcontext[140];
	memset(mcontext, 0, 140);
	memcpy(mcontext, header, header_len);
	memcpy(mcontext + header_len, nonce, nonce_len);

	blake2b_state_t initialCtx;
	zcash_blake2b_init(&initialCtx, ZCASH_HASH_LEN, PARAM_N, PARAM_K);
	zcash_blake2b_update(&initialCtx, (const uint8_t*)mcontext, 128, 0);

	uint64_t blake_iv[] =
	{
		0x6a09e667f3bcc908, 0xbb67ae8584caa73b,
		0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
		0x510e527fade682d1, 0x9b05688c2b3e6c1f,
		0x1f83d9abfb41bd6b, 0x5be0cd19137e2179,
	};

	memcpy(&blake_data[0], initialCtx.h, 64);
	memcpy(&blake_data[8], blake_iv, 64);
	
	blake_data[12] ^= 144;
	blake_data[14] ^= ~0ULL;
	
	checkCudaErrors(hipMemcpy(ctx.d_blake_data, blake_data, 128, hipMemcpyHostToDevice));

	//test<16><<<4096, 256>>>(ctx.d_data->blake);
	
	kernel_round0<<<4096, 256>>>(ctx.d_data, ctx.d_blake_data);
	//PrintAverageRowCount(1, ctx);
	kernel_round1<<<4096, 608 >>>(ctx.d_data);
	//PrintAverageBinCount(1, ctx);
	//PrintAverageRowCount(2, ctx);
	kernel_round2<<<4096, 608 >>>(ctx.d_data);
	//PrintAverageBinCount(2, ctx);
	//PrintAverageRowCount(3, ctx);
	kernel_round3<<<4096, 608>>>(ctx.d_data);
	//PrintAverageBinCount(3, ctx);
	//PrintAverageRowCount(4, ctx);
	kernel_round4<<<4096, 608>>>(ctx.d_data);
	//PrintAverageBinCount(4, ctx);
	//PrintAverageRowCount(5, ctx);
	kernel_round5<<<4096, 608 >>>(ctx.d_data);
	//PrintAverageBinCount(5, ctx);
	//PrintAverageRowCount(6, ctx);
	kernel_round6<<<4096, 608 >>>(ctx.d_data);
	//PrintAverageBinCount(6, ctx);
	//PrintAverageRowCount(7, ctx);
	kernel_round7<<<4096, 608 >>>(ctx.d_data);
	//PrintAverageBinCount(7, ctx);
	//PrintAverageRowCount(8, ctx);
	kernel_round8<<<4096, 608 >>>(ctx.d_data);
	//PrintAverageBinCount(8, ctx);
	//PrintAverageRowCount(9, ctx);
	kernel_round9<<<4096, 608 >>>(ctx.d_data);
	kernel_candidates<<<512, 128>>>(ctx.d_data);

	checkCudaErrors(hipMemcpy(ctx.h_candidates, &ctx.d_data->candidates, sizeof(candidate_t), hipMemcpyDeviceToHost));
	
	ctx.h_candidates->sol_nr[0] = min(16, ctx.h_candidates->sol_nr[0]);

	//uint8_t valid[16] = { 0 };
	//for (unsigned sol_i = 0; sol_i < ctx.h_candidates->sol_nr[0]; sol_i++) {
	//	verify_sol(ctx.h_candidates, sol_i, valid);
	//}

	int sols_found = 0;
	uint8_t proof[COMPRESSED_PROOFSIZE * 2];
	for (uint32_t i = 0; i < ctx.h_candidates->sol_nr[0]; i++) {
		//if (valid[i]) {
			compress(proof, (uint32_t *)(ctx.h_candidates->vals[i]), 1 << PARAM_K);
			speed.AddSolution();
			sols_found++;
		//}
	}
}

using stratum::primitives::CBlock;
using stratum::primitives::CEquihashInput;
using stratum::CDataStream;
using stratum::arith::uint256;

static std::vector<uint256*> benchmark_nonces;

static void generate_nounces(int hashes)
{
	std::srand(std::time(0));
	benchmark_nonces.push_back(new uint256());
	benchmark_nonces.back()->begin()[31] = 1;
	for (int i = 0; i < (hashes - 1); ++i)
	{
		benchmark_nonces.push_back(new uint256());
		for (unsigned int i = 0; i < 32; ++i)
			benchmark_nonces.back()->begin()[i] = std::rand() % 256;
	}
}

context g_ctx;

static bool benchmark_solve(context& ctx, const CBlock& block, const char* header, unsigned int header_len)
{
	if (benchmark_nonces.empty()) {
		return false;
	}
	
	uint256* nonce = benchmark_nonces.front();
	benchmark_nonces.erase(benchmark_nonces.begin());
	
	solve(ctx, header, header_len, (const char*)nonce->begin(), nonce->size());
	
	std::fill(&bins[0], &bins[256], 0);

	delete nonce;
	
	return true;
}

static int benchmark()
{
	try
	{
		CBlock pblock;
		CEquihashInput I{ pblock };
		CDataStream ss(stratum::SER_NETWORK, PROTOCOL_VERSION);
		ss << I;

		const char *tequihash_header = (char *)&ss[0];
		unsigned int tequihash_header_len = ss.size();

		while(benchmark_solve(g_ctx, pblock, tequihash_header, tequihash_header_len));
	}
	catch (const std::runtime_error &e)
	{
		exit(0);
		return 0;
	}
	
	return 0;
}


#include <thread>
#include <atomic>
int main()
{
	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

	g_ctx.init();

	//Step 1 - Generate nouces
	generate_nounces(10000);

	std::atomic<int> amdone(0);

	std::thread tellme([&amdone]() {
		for (; amdone.load() == 0;) {
			std::this_thread::sleep_for(std::chrono::seconds(2));
			std::cout << speed.GetSolutionSpeed() << " Sols/s" << std::endl;
		}
	});
	
	benchmark();
	
	printf("final %.2f sols/s\n", speed.GetSolutionSpeed());
	
	amdone.store(1);
	tellme.join();

	g_ctx.destroy();

	hipDeviceReset();

	return 0;
}
